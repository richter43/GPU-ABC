#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include "utils.h"

//Host utility functions
void *malloc_errorhandler(size_t size){
	void *tmp_malloc = malloc(size);
	if(tmp_malloc == NULL){
		fprintf(stderr, "Error, could not allocate memory in host device\n");
		exit(EXIT_FAILURE);
	}
	return tmp_malloc;
}

hiprandState *create_random_state(int blocks, int threads, int seed){

	hiprandState *state;
	checkCudaErrors(hipMalloc(&state, sizeof(hiprandState)*blocks*threads));
        setup_kernel<<<blocks, threads>>>(state, seed);

	return state;
}

float *create_sol_array(int rows, int cols){
	float *d_array;
	checkCudaErrors(hipMalloc(&d_array, sizeof(float)*rows*cols));
	return d_array;
}

//Device utility functions
__global__ void setup_kernel(hiprandState *state, int seed){
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	//id could be used as seed to generate a different pseudorandom sequence
	hiprand_init(seed, id, 0, &state[id]);
	return;
}

__device__ float get_random_float(hiprandState *state, float min, float max){
	float val = hiprand_uniform(state);
	val = min + val*(max-min);
	return val;
}

__device__ void random_float_array(hiprandState *state, float *ret, int size, float min, float max){
	for(int i = 0; i < size; i++){
		ret[i] = get_random_float(state, min, max);
	}
	return;
}
