#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

__global__ void setup_kernel(hiprandState *state, int seed){
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	//id could be used as seed to generate a different pseudorandom sequence
	hiprand_init(seed, id, 0, &state[id]);
	return;
}

__device__ float get_random_float(hiprandState *state, float min, float max){
	//Test pending
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	float val = hiprand_uniform(&state[id]);
	val = min + val*(max-min);
	return val;
}

__device__ void random_float_array(hiprandState *state, float *ret, int size, float min, float max){
	for(int i = 0; i < size; i++){
		ret[i] = get_random_float(state, min, max);
	}
	return;
}

hiprandState *create_random_state(int blocks, int threads, int seed){

	hiprandState *state;
	checkCudaErrors(hipMalloc(&state, sizeof(hiprandState)*blocks*threads));
        setup_kernel<<<blocks, threads>>>(state, seed);

	return state;
}

float *create_sol_array(int rows, int cols){
	float *d_array;
	checkCudaErrors(hipMalloc(&d_array, sizeof(float)*rows*cols));
	return d_array;
}
