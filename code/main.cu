#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>

#include "main.h"
#include "abc.h"
#include "benchfuns.h"
#include "utils.h"

//Bee state array

int main(int argc, char *argv[]){

	//Handling passed arguments
	if(argc != 4){
		fprintf(stderr, "Missing parameters (Iterations, ratio of onlooker to employed bees, max patience)\n");
		exit(EXIT_FAILURE);
	}

	int iterations;
	float ratio_ote;
	int max_patience;

	if(sscanf(argv[1], "%d", &iterations)!= 1 ||  sscanf(argv[2], "%f", &ratio_ote) != 1 || sscanf (argv[3], "%d", &max_patience) != 1){
                fprintf(stderr, "Error during parsing of the passed parameters\n");
		exit(EXIT_FAILURE);
        }

	if(ratio_ote < 0.0 || ratio_ote > 1.0){
		fprintf(stderr, "Ratio should be a number between 0.0 and 1.0\n");
		exit(EXIT_FAILURE);
	}

	//Host code
	float *h_best_sol_fitness; //Contains both best solution and its fitness
	h_best_sol_fitness = (float *) malloc_errorhandler(sizeof(float)*BLOCKS*(DIM+1));

	//Device code
	//Creation of a random state
	hiprandState *d_state = create_random_state(BLOCKS, THREADS, SEED);
	//Device memory allocation
	float *d_solutions, *d_fitness, *d_best_sol_fitness;
	checkCudaErrors(hipMalloc(&d_solutions, sizeof(float)*BLOCKS*THREADS*DIM));
	checkCudaErrors(hipMalloc(&d_fitness, sizeof(float)*BLOCKS*THREADS));
	checkCudaErrors(hipMalloc(&d_best_sol_fitness, sizeof(float)*BLOCKS*(DIM+1)));

	//Struct that contains all the relevant addresses and information
	abc_info_t h_container = { d_state, d_solutions, d_best_sol_fitness, d_fitness, BLOCKS*THREADS*DIM, DIM, MIN_FLOAT, MAX_FLOAT, iterations, max_patience, ratio_ote};
	//Kernel execution
	#if TEST_CONSTANT
	copy_container_symbol(&h_container);
	abc_algo<<<BLOCKS,THREADS>>>();
	#else
	abc_algo<<<BLOCKS,THREADS>>>(h_container);
	#endif

	checkCudaErrors(hipDeviceSynchronize());
	
	checkCudaErrors(hipMemcpy(h_best_sol_fitness, d_best_sol_fitness, sizeof(float)*BLOCKS*(DIM+1), hipMemcpyDeviceToHost));

	float *tmp_sol = (float*) calloc(DIM,sizeof(float));
	if(tmp_sol == NULL){
		fprintf(stderr, "Could not allocate memory\n");
		exit(EXIT_FAILURE);
	}

	//Printing solutions
	for(int i = 0; i < BLOCKS; i++){
		printf("Hive %d solution: ", i);
		for(int j = 0; j < DIM; j++){
			tmp_sol[j] += h_best_sol_fitness[j + (DIM+1)*i]; 
			printf("%f ", h_best_sol_fitness[j + (DIM+1)*i]);
		}
		printf("\nBest fitness: %f\n", h_best_sol_fitness[DIM + (DIM+1)*i]);
	}

	//Wisdom of crowds principle
	printf("Wisdom of crowds solution: \n");
	for(int i = 0; i < DIM; i++){
		tmp_sol[i] = tmp_sol[i] / BLOCKS;
		printf("%f ", tmp_sol[i]);
	}

	free(h_best_sol_fitness);
	hipFree(d_solutions);	
	hipFree(d_state);	
	hipFree(d_fitness);
	hipFree(d_best_sol_fitness);

	return EXIT_SUCCESS;
}
