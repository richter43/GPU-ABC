#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include "utils.h"

__device__ void scout_bee(hiprandState *state, float *sol, int dim, float min, float max){
	random_float_array(state, sol, dim, min, max);	
	return;
}
